#include "hip/hip_runtime.h"
// Can not use std::stl within cuda kernels

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <vector>
#include <iostream>
#include <typeinfo>

using namespace std;

class vector3d
{
    public: 
        float x, y, z;

        vector3d()
        {
            x = 0;
            y = 0;
            z = 0;
        }

        vector3d(float x_, float y_, float z_)
        {
            x = x_;
            y = y_;
            z = z_;
        }

        inline vector3d& operator +=(const vector3d& rhs)
        {
            x += rhs.x;
            y += rhs.y;
            z += rhs.z;
            return *this;
        }

        inline vector3d& operator +(const vector3d& rhs)
        {
            *this += rhs;
            return *this;
        }

        inline vector3d& operator /=(const float& rhs)
        {
            x /= rhs;
            y /= rhs;
            z /= rhs;
            return *this;
        }

        inline vector3d& operator /(const float& rhs)
        {
            *this /= rhs;
            return *this;
        }

        friend ostream &operator<<( ostream &output, const vector3d &D ) { 
            output << D.x << ","<<D.y<<","<<D.z<<endl;
            return output;            
         }

        __host__ __device__ inline void test()
        {
            int test = 0;
            test++;
        }
        
};

__global__ void vecAdd(vector<vector3d> a, vector<vector3d> b, vector<vector3d> c, int n)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;

    if (id < n)
    {
        c[id].x = a[id].x + b[id].x;
        c[id].y = a[id].y + b[id].y;
        c[id].z = a[id].z + b[id].z;

        c[id].test();
    }
}
 
int main( int argc, char* argv[] )
{
    // Size of vectors
    int n = 1000000;
    float n_f = 1000000.0f;
 		
    // Host input vectors
    vector<vector3d> h_a;
    vector<vector3d> h_b;
    
    //Host output vector
    vector<vector3d> h_c;
 
    // Device input vectors
    vector<vector3d> *d_a;
    vector<vector3d> *d_b;

    //Device output vector
    vector<vector3d> *d_c;
 
    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(vector<vector3d>);
 
    // Allocate memory for each vector on host
    h_a.reserve(n);
    h_b.reserve(n);
    h_c.reserve(n);
 
    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
 
    // Initialize vectors on host
    for(int i = 0; i < n; i++ ) {
        h_a[i] = vector3d(1,2,3);
        h_b[i] = vector3d(1,2,3);
    }

    // // Copy host vectors to device
    hipMemcpy( d_a, &h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, &h_b, bytes, hipMemcpyHostToDevice);
 
    // int blockSize, gridSize;
 
    // // Number of threads in each thread block
    // blockSize = 1024;
 
    // // Number of thread blocks in grid
    // gridSize = (int)ceil((float)n/blockSize);
 
    // // Execute the kernel
    // vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    // // Copy array back to host
    // hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
 
    // // Sum up vector c and print result divided by n, this should equal 1 within error
    // vector3d sum;
    // for(int i=0; i<=n; i++)
    //     sum += h_c[i];

    // printf("final result: %f, %f, %f\n", sum.x / n_f, sum.y / n_f, sum.z / n_f);
    // printf("final result: %f, %f, %f\n", sum.x, sum.y, sum.z);
 
    // // Release device memory
    // hipFree(d_a);
    // hipFree(d_b);
    // hipFree(d_c);
 
    // Release host memory
    // free(h_a);
    // free(h_b);
    // free(h_c);
 
    return 0;
}
